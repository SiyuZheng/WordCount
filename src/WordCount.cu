#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <string>
#include <cstdio>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "WordCount.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include ""

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

void cudaMap(char *input, KeyValuePair *pairs) {
	kernMap << < GRID_SIZE, BLOCK_SIZE >> >(input, pairs);
	checkCUDAError("Map kernel failed!");
	hipDeviceSynchronize();
}

__global__ void kernMap(char *idata, KeyValuePair *pairs) {
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = ind; i < NUM_INPUT; i += offset) {
		mapper(&idata[i], &pairs[i * NUM_KEYS]);
	}
}

__device__ void mapper(char *input, KeyValuePair *pairs)
{
	pairs->key = 0;
	char ch = *input;
	if (ch == ' ' || ch == '\n')
	{
		pairs->value = 1;
	}
	else
	{
		pairs->value = 0;
	}
}


void cudaReduce(KeyValuePair *pairs, int *odata) {
	kernReduce << <GRID_SIZE, BLOCK_SIZE >> >(pairs, odata);
	checkCUDAError("Reduce kernel failed!");
	hipDeviceSynchronize();
}

__global__ void kernReduce(KeyValuePair *pairs, int *odata) {
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = ind; i < NUM_OUTPUT; i += offset) {
		int startIndex = 0;
		int count = 0;
		int valueSize = 0;
		int j;

		for (j = 1; j < NUM_INPUT * NUM_KEYS; j++) {
			if (KVComparator()(pairs[j - 1], pairs[j])) {
				if (count == i) {
					// This thread has found the correct number
					// There is a bit of warp divergence here as some threads
					// break before others, but we still make the most out of it
					// by calling the reducer at the very end, so there is not
					// any warp divergence where the bulk of the computation
					// should occur (the reducer).
					break;
				}
				else {
					count++;
					startIndex = j;
				}
			}
		}
		valueSize = j - startIndex;
		reducer(pairs + startIndex, valueSize, &odata[i]);
	}
}

__device__ void reducer(KeyValuePair *pairs, int len, int* odata)
{
	int sum = 0;
	for (KeyValuePair *pair = pairs; pair != pairs + len; pair++)
	{
		sum += pair->value;
	}
	*odata = sum;
}

void cudaMapReduce(char* input, int *output) {
	char* dev_idata;
	int* dev_odata;
	KeyValuePair *dev_intermediate;

	hipMalloc(&dev_idata, NUM_INPUT * sizeof(char));
	hipMalloc(&dev_intermediate, NUM_INPUT * NUM_KEYS * sizeof(KeyValuePair));
	hipMalloc(&dev_odata, NUM_OUTPUT * sizeof(int));

	hipMemcpy(dev_idata, input, NUM_INPUT * sizeof(char), hipMemcpyHostToDevice);

	cudaMap(dev_idata, dev_intermediate);

	thrust::device_ptr<KeyValuePair> dev_ptr(dev_intermediate);
	thrust::sort(dev_ptr, dev_ptr + NUM_INPUT * NUM_KEYS, KVComparator());

	cudaReduce(dev_intermediate, dev_odata);

	hipMemcpy(output, dev_odata, NUM_OUTPUT * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_intermediate);
	hipFree(dev_odata);
}

const int N = 20;
const int blocksize = 20;

__global__
void hello(char *a, int *c, int size)
{
	int i = 0, k = 0;
	int count = 1;

	if (threadIdx.x < size) {

		for (i = 0; i < size; i++) {
			if (threadIdx.x != i) {
				for (k = 0; k < N; k++) {
					if (a[N * threadIdx.x + k] != a[N * i + k]) {
						break;
					}
					if (k == N - 1) {
						count++;
					}
				}
			}
		}

		c[threadIdx.x] = count;
	}
	else {
		c[threadIdx.x] = -1;
	}
}

int main(int argc, char* argv[]) {
	char* idata = new char[NUM_INPUT];
	int* odata = new int[NUM_OUTPUT];
	FILE* fp;
	fp = fopen("hamlet.txt", "r");
	int i = 0;
	int ch;
	while (1) {
		if (fp == NULL) {
			printf("File didn't open");
			break;
		}
		ch = fgetc(fp);
		if (ch == EOF) {
			break;
		}
		i++;
		idata[i] = ch;
		printf("%c", ch);
	}
	if (fp != NULL) {
		fclose(fp);
	}
	cudaMapReduce(idata, odata);
	for (int i = 0; i < NUM_OUTPUT; i++)
	{
		printf("The total number of words in the file are: %d\n", odata[i]);
	}
	delete idata;
	delete odata;
	return 0;
	//char words[N][N] =
	//{
	//	"ABCDE",
	//	"xyz",
	//	"Hi",
	//	"japan",
	//	"xyz",
	//	"Hi",
	//	"cup",
	//	"paper",
	//	"Hi",
	//	"Apple"
	//};

	//int size = 10;

	//int count[N];

	//char answer_words[N][N];
	//int answer_count[N];

	//char *ad;
	//int *cd;

	//const int csize = N*N * sizeof(char);
	//const int isize = N * sizeof(int);

	//hipMalloc((void**)&ad, csize);
	//hipMalloc((void**)&cd, isize);

	//hipMemcpy(ad, words, csize, hipMemcpyHostToDevice);

	//dim3 dimBlock(blocksize, 1);
	//dim3 dimGrid(1, 1);
	//hello << <dimGrid, dimBlock >> >(ad, cd, size);
	//hipMemcpy(count, cd, isize, hipMemcpyDeviceToHost);
	//hipFree(ad);
	//hipFree(cd);

	//int i = 0, k = 0;

	//int num = 0;
	//int dismatchflag = 0;

	//for (i = 0; i < N; i++) {
	//	if (count[i] == -1) {
	//		break;
	//	}

	//	if (count[i] == 1) {
	//		strcpy(answer_words[num], words[i]);
	//		answer_count[num] = count[i];
	//		num++;
	//	}
	//	else if (count[i] > 1) {
	//		for (k = 0; k < num; k++) {
	//			if (strcmp(words[i], answer_words[k]) == 0) {
	//				dismatchflag = 1;
	//				break;
	//			}
	//		}
	//		if (dismatchflag == 0) {
	//			strcpy(answer_words[num], words[i]);
	//			answer_count[num] = count[i];
	//			num++;
	//		}
	//		else {
	//			dismatchflag = 0;
	//		}
	//	}
	//}

	//for (i = 0; i < num; i++) {
	//	printf("%s, %d\n", answer_words[i], answer_count[i]);
	//}

	//return EXIT_SUCCESS;
}