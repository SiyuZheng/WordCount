#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <stdlib.h>
#include <string>
#include <cstdio>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "WordCount.h"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include ""
#include <sstream>
#include <string>
#include <fstream>
#include <tchar.h>
#include "string"

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)

void cudaMap(char *input, KeyValuePair *pairs) {
	kernMap << < GRID_SIZE, BLOCK_SIZE >> >(input, pairs);
	checkCUDAError("Map kernel failed!");
	hipDeviceSynchronize();
}

__global__ void kernMap(char *idata, KeyValuePair *pairs) {
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = ind; i < NUM_INPUT; i += offset) {
		mapper(&idata[i], &pairs[i * NUM_KEYS]);
	}
}

__device__ void mapper(char *input, KeyValuePair *pairs)
{
	pairs->key = 0;
	char ch = *input;
	if (ch == ' ' || ch == '\n' || ch == ',' || ch == '.')
	{
		pairs->value = 1;
	}
	else
	{
		pairs->value = 0;
	}
	
}


void cudaReduce(KeyValuePair *pairs, int *odata) {
	kernReduce << <GRID_SIZE, BLOCK_SIZE >> >(pairs, odata);
	checkCUDAError("Reduce kernel failed!");
	hipDeviceSynchronize();
}

__global__ void kernReduce(KeyValuePair *pairs, int *odata) {
	int ind = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;
	for (int i = ind; i < NUM_OUTPUT; i += offset) {
		int startIndex = 0;
		int count = 0;
		int valueSize = 0;
		int j;

		for (j = 1; j < NUM_INPUT * NUM_KEYS; j++) {
			if (KVComparator()(pairs[j - 1], pairs[j])) {
				if (count == i) {
					break;
				}
				else {
					count++;
					startIndex = j;
				}
			}
		}
		valueSize = j - startIndex;
		reducer(pairs + startIndex, valueSize, &odata[i]);
	}
}

__device__ void reducer(KeyValuePair *pairs, int len, int* odata)
{
	int sum = 0;
	for (KeyValuePair *pair = pairs; pair != pairs + len; pair++)
	{
		sum += pair->value;
	}
	*odata = sum;
}

void cudaMapReduce(char* input, int *output) {
	char* dev_idata;
	int* dev_odata;
	KeyValuePair *dev_intermediate;

	hipMalloc(&dev_idata, NUM_INPUT * sizeof(char));
	hipMalloc(&dev_intermediate, NUM_INPUT * NUM_KEYS * sizeof(KeyValuePair));
	hipMalloc(&dev_odata, NUM_OUTPUT * sizeof(int));

	hipMemcpy(dev_idata, input, NUM_INPUT * sizeof(char), hipMemcpyHostToDevice);

	cudaMap(dev_idata, dev_intermediate);

	thrust::device_ptr<KeyValuePair> dev_ptr(dev_intermediate);
	thrust::sort(dev_ptr, dev_ptr + NUM_INPUT * NUM_KEYS, KVComparator());

	cudaReduce(dev_intermediate, dev_odata);

	hipMemcpy(output, dev_odata, NUM_OUTPUT * sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_idata);
	hipFree(dev_intermediate);
	hipFree(dev_odata);
}


int chars = 0, words = 0, lines = 1;
char c;

void CPUCounting(FILE *file)
{
	while ((c = fgetc(file)) != EOF)
	{
		chars++;
		if (c != ' '&& c != ',' && c != '\n')
		{
			words++;
			while ((c = fgetc(file)) != EOF)
			{
				chars++;
				if (c != ' '&& c != ','&& c != '\n')
				{
				}
				else if (c == '\n')
				{

					lines++;
					break;
				}
				else if (c == ' ' || c == ',' || c == '\n')
					break;
				else
				{
					break;
				}
			}
		}

		else if (c == '\n')
		{
			lines++;
		}
	}
}

int main(int argc, char* argv[]) {
	//// read by line
	//std::ifstream infile("hamlet.txt");
	//std::string line;
	//if (infile) {
	//	while (std::getline(infile, line)) {
	//		std::cout << line << std::endl;
	//	}
	//}
	//else {
	//	std::cout << "no such file" << std::endl;
	//}

	char* idata = new char[NUM_INPUT];
	int* odata = new int[NUM_OUTPUT];
	char* filename = "test.txt";
	FILE* fp;
	fp = fopen(filename, "r");
	int i = 0;
	int ch;
	while (1) {
		if (fp == NULL) {
			printf("File didn't open");
			break;
		}
		ch = fgetc(fp);
		if (ch == EOF) {
			break;
		}
		i++;
		idata[i] = ch;
		printf("%c", ch);
	}
	FILE* fp2;
	fp2 = fopen(filename, "r");
	CPUCounting(fp2);
	if (fp != NULL) {
		fclose(fp);
	}
	if (fp2 != NULL) {
		fclose(fp);
	}
	cudaMapReduce(idata, odata);

	for (int i = 0; i < argc; i++)
	{
		std::cout << "CPU computing: " << std::endl;
		std::cout << "Total word count: " << words << std::endl;
	}
	for (int i = 0; i < NUM_OUTPUT; i++)
	{
		std::cout << "GPU computing: " << std::endl;
		std::cout << "Total word count: " << odata[i] << std::endl;
	}
	delete idata;
	delete odata;
	return 0;
}